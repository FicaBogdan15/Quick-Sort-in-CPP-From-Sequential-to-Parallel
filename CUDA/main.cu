#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>

__global__ void partitionPhase1(int* d_data,
                                int* d_bufSmall,
                                int* d_bufLarge,
                                int* d_countSmall,
                                int* d_countLarge,
                                int n,
                                int pivot)
{
    int tid = threadIdx.x;
    int quarter = (n+3) / 4;
    int start   = tid * quarter;
    int end     = min(start + quarter, n);

    int sc = 0, lc = 0;
    for(int i = start; i < end; i++) {
        int v = d_data[i];
        if (v < pivot) {
            d_bufSmall[tid * quarter + sc++] = v;
        } else {
            d_bufLarge[tid * quarter + lc++] = v;
        }
    }
    d_countSmall[tid] = sc;
    d_countLarge[tid] = lc;
}

__global__ void partitionPhase2(int* d_bufSmall,
                                int* d_bufLarge,
                                int* d_dataOut,
                                int* d_countSmall,
                                int* d_countLarge,
                                int n)
{
    int tid     = threadIdx.x;     // 0..3
    int quarter = (n+3) / 4;

    int totalSmall = 0;
    #pragma unroll
    for(int i = 0; i < 4; i++)
        totalSmall += d_countSmall[i];

    int sources[2], cnts[2], offset = 0;
    if (tid == 0) {
        // A preia < pivot din A(0) apoi din D(3)
        sources[0] = 0; sources[1] = 3;
        cnts[0]    = d_countSmall[0];
        cnts[1]    = d_countSmall[3];
        offset     = 0;
        for(int s = 0; s < 2; s++) {
            int src = sources[s], cnt = cnts[s];
            for(int i = 0; i < cnt; i++)
                d_dataOut[offset + i] = d_bufSmall[src * quarter + i];
            offset += cnt;
        }
    }
    else if (tid == 1) {
        // B preia < pivot din B(1) apoi din C(2)
        sources[0] = 1; sources[1] = 2;
        cnts[0]    = d_countSmall[1];
        cnts[1]    = d_countSmall[2];
        offset     = d_countSmall[0] + d_countSmall[3];
        for(int s = 0; s < 2; s++) {
            int src = sources[s], cnt = cnts[s];
            for(int i = 0; i < cnt; i++)
                d_dataOut[offset + i] = d_bufSmall[src * quarter + i];
            offset += cnt;
        }
    }
    else if (tid == 2) {
        // C preia >= pivot din C(2) apoi din B(1)
        sources[0] = 2; sources[1] = 1;
        cnts[0]    = d_countLarge[2];
        cnts[1]    = d_countLarge[1];
        offset     = totalSmall;
        for(int s = 0; s < 2; s++) {
            int src = sources[s], cnt = cnts[s];
            for(int i = 0; i < cnt; i++)
                d_dataOut[offset + i] = d_bufLarge[src * quarter + i];
            offset += cnt;
        }
    }
    else if (tid == 3) {
        // D preia >= pivot din D(3) apoi din A(0)
        sources[0] = 3; sources[1] = 0;
        cnts[0]    = d_countLarge[3];
        cnts[1]    = d_countLarge[0];
        offset     = totalSmall + d_countLarge[2] + d_countLarge[1];
        for(int s = 0; s < 2; s++) {
            int src = sources[s], cnt = cnts[s];
            for(int i = 0; i < cnt; i++)
                d_dataOut[offset + i] = d_bufLarge[src * quarter + i];
            offset += cnt;
        }
    }
}

void quickSortCUDA(int* h_data, int n)
{
    int *d_data, *d_bufSmall, *d_bufLarge, *d_dataOut;
    int *d_countSmall, *d_countLarge;
    hipMalloc(&d_data,       n * sizeof(int));
    hipMalloc(&d_bufSmall,   n * sizeof(int));
    hipMalloc(&d_bufLarge,   n * sizeof(int));
    hipMalloc(&d_dataOut,    n * sizeof(int));
    hipMalloc(&d_countSmall, 4 * sizeof(int));
    hipMalloc(&d_countLarge, 4 * sizeof(int));

    hipMemcpy(d_data, h_data, n * sizeof(int), hipMemcpyHostToDevice);

    int pivot = h_data[0];

    partitionPhase1<<<1,4>>>(d_data,
                             d_bufSmall,
                             d_bufLarge,
                             d_countSmall,
                             d_countLarge,
                             n,
                             pivot);
    hipDeviceSynchronize();

    partitionPhase2<<<1,4>>>(d_bufSmall,
                             d_bufLarge,
                             d_dataOut,
                             d_countSmall,
                             d_countLarge,
                             n);
    hipDeviceSynchronize();

    hipMemcpy(h_data, d_dataOut, n * sizeof(int), hipMemcpyDeviceToHost);

    int countSmall[4];
    hipMemcpy(countSmall, d_countSmall, 4*sizeof(int), hipMemcpyDeviceToHost);
    int totalSmall = countSmall[0] + countSmall[1]
                   + countSmall[2] + countSmall[3];

    std::sort(h_data,               h_data + totalSmall);
    std::sort(h_data + totalSmall,  h_data + n);

    hipFree(d_data);
    hipFree(d_bufSmall);
    hipFree(d_bufLarge);
    hipFree(d_dataOut);
    hipFree(d_countSmall);
    hipFree(d_countLarge);
}

int main()
{
    std::cout << "Opening input file...\n";
    std::ifstream fin(R"(C:\Users\rollo\OneDrive\Desktop\inputs\input12M.txt)");
    if (!fin) {
        std::cerr << "Error: could not open input file\n";
        return 1;
    }

    std::cout << "Creating output file...\n";
    std::ofstream fout("output.txt");
    if (!fout) {
        std::cerr << "Error: could not create output file\n";
        return 1;
    }

    std::cout << "Reading data into vector...\n";
    std::vector<int> h_vec;
    int x;
    while (fin >> x) {
        h_vec.push_back(x);
    }
    fin.close();
    std::cout << "Read " << h_vec.size() << " elements\n";


    int N = static_cast<int>(h_vec.size());
    int* h_arr = new int[N];
    for (int i = 0; i < N; i++)
        h_arr[i] = h_vec[i];

    std::cout << "Launching QuickSort on GPU (" << N << " elements)...\n";
    auto t0 = std::chrono::high_resolution_clock::now();
    quickSortCUDA(h_arr, N);
    auto t1 = std::chrono::high_resolution_clock::now();

    double elapsed_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
    std::cout << "QuickSort CUDA completed in "
              << elapsed_ms << " ms\n";

    std::cout << "Writing sorted data to output.txt...\n";
    for (int i = 0; i < N; i++) {
        fout << h_arr[i];
        if (i + 1 < N) fout << ' ';
    }
    fout << '\n';
    fout.close();

    delete[] h_arr;
    std::cout << "Done.\n";
    return 0;
}
